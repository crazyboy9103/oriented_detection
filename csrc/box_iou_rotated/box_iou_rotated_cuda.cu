#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include "box_iou_rotated_utils.h"


namespace detectron2 {

// 2D block with 32 * 16 = 512 threads per block
const int BLOCK_DIM_X = 32;
const int BLOCK_DIM_Y = 16;

template <typename T>
__global__ void box_iou_rotated_cuda_kernel(
    const int n_boxes1,
    const int n_boxes2,
    const T* dev_boxes1,
    const T* dev_boxes2,
    T* dev_ious, 
    const bool aligned) {
  if (aligned) {
    CUDA_1D_KERNEL_LOOP(index, n_boxes1) {
      int b1 = index;
      int b2 = index;

      int base1 = b1 * 5;

      float block_boxes1[5];
      float block_boxes2[5];

      block_boxes1[0] = dev_boxes1[base1 + 0];
      block_boxes1[1] = dev_boxes1[base1 + 1];
      block_boxes1[2] = dev_boxes1[base1 + 2];
      block_boxes1[3] = dev_boxes1[base1 + 3];
      block_boxes1[4] = dev_boxes1[base1 + 4];

      int base2 = b2 * 5;

      block_boxes2[0] = dev_boxes2[base2 + 0];
      block_boxes2[1] = dev_boxes2[base2 + 1];
      block_boxes2[2] = dev_boxes2[base2 + 2];
      block_boxes2[3] = dev_boxes2[base2 + 3];
      block_boxes2[4] = dev_boxes2[base2 + 4];

      dev_ious[index] =
          single_box_iou_rotated<T>(block_boxes1, block_boxes2);
    }
  } else {
    CUDA_1D_KERNEL_LOOP(index, n_boxes1 * n_boxes2) {
      int b1 = index / n_boxes2;
      int b2 = index % n_boxes2;

      int base1 = b1 * 5;

      float block_boxes1[5];
      float block_boxes2[5];

      block_boxes1[0] = dev_boxes1[base1 + 0];
      block_boxes1[1] = dev_boxes1[base1 + 1];
      block_boxes1[2] = dev_boxes1[base1 + 2];
      block_boxes1[3] = dev_boxes1[base1 + 3];
      block_boxes1[4] = dev_boxes1[base1 + 4];

      int base2 = b2 * 5;

      block_boxes2[0] = dev_boxes2[base2 + 0];
      block_boxes2[1] = dev_boxes2[base2 + 1];
      block_boxes2[2] = dev_boxes2[base2 + 2];
      block_boxes2[3] = dev_boxes2[base2 + 3];
      block_boxes2[4] = dev_boxes2[base2 + 4];

      dev_ious[index] =
          single_box_iou_rotated<T>(block_boxes1, block_boxes2);
      }
  }
}

at::Tensor box_iou_rotated_cuda(
    // input must be contiguous
    const at::Tensor& boxes1,
    const at::Tensor& boxes2) {
  using scalar_t = float;
  AT_ASSERTM(boxes1.scalar_type() == at::kFloat, "boxes1 must be a float tensor");
  AT_ASSERTM(boxes2.scalar_type() == at::kFloat, "boxes2 must be a float tensor");
  AT_ASSERTM(boxes1.is_cuda(), "boxes1 must be a CUDA tensor");
  AT_ASSERTM(boxes2.is_cuda(), "boxes2 must be a CUDA tensor");
  at::cuda::CUDAGuard device_guard(boxes1.device());

  auto num_boxes1 = boxes1.size(0);
  auto num_boxes2 = boxes2.size(0);

  at::Tensor ious =
      at::empty({num_boxes1 * num_boxes2}, boxes1.options().dtype(at::kFloat));
  auto output_size = ious.numel();

  bool transpose = false;
  if (num_boxes1 > 0 && num_boxes2 > 0) {
    scalar_t *data1 = boxes1.data_ptr<scalar_t>(),
             *data2 = boxes2.data_ptr<scalar_t>();

    if (num_boxes2 > 65535 * BLOCK_DIM_Y) {
      AT_ASSERTM(
          num_boxes1 <= 65535 * BLOCK_DIM_Y,
          "Too many boxes for box_iou_rotated_cuda!");
      // x dim is allowed to be large, but y dim cannot,
      // so we transpose the two to avoid "invalid configuration argument"
      // error. We assume one of them is small. Otherwise the result is hard to
      // fit in memory anyway.
      std::swap(num_boxes1, num_boxes2);
      std::swap(data1, data2);
      transpose = true;
    }

    // const int blocks_x =
    //     at::cuda::ATenCeilDiv(static_cast<int>(num_boxes1), BLOCK_DIM_X);
    // const int blocks_y =
    //     at::cuda::ATenCeilDiv(static_cast<int>(num_boxes2), BLOCK_DIM_Y);

    // dim3 blocks(blocks_x, blocks_y);
    // dim3 threads(BLOCK_DIM_X, BLOCK_DIM_Y);
    at::cuda::CUDAGuard device_guard(boxes1.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // box_iou_rotated_cuda_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
    //     num_boxes1,
    //     num_boxes2,
    //     data1,
    //     data2,
    //     (scalar_t*)ious.data_ptr<scalar_t>());
    bool aligned = true;
    box_iou_rotated_cuda_kernel<scalar_t>
      <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
          num_boxes1, num_boxes2, boxes1.data_ptr<scalar_t>(),
          boxes2.data_ptr<scalar_t>(), (scalar_t*)ious.data_ptr<scalar_t>(),
          aligned);
    AT_CUDA_CHECK(hipGetLastError());
  }

  // reshape from 1d array to 2d array
  auto shape = std::vector<int64_t>{num_boxes1, num_boxes2};
  if (transpose) {
    return ious.view(shape).t();
  } else {
    return ious.view(shape);
  }
}

} // namespace detectron2
