// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved
// modified from
// https://github.com/facebookresearch/detectron2/blob/master/detectron2/layers/csrc/box_iou_rotated/box_iou_rotated_cuda.cu
#include "box_iou_rotated_cuda.cuh"
#include "../pytorch_cuda_helper.hpp"

namespace mmrotate {
    at::Tensor box_iou_rotated_cuda(
        const Tensor& boxes1, 
        const Tensor& boxes2, 
        const int mode_flag, 
        const bool aligned
    ) {
    using scalar_t = float;
    TORCH_INTERNAL_ASSERT(boxes1.is_cuda(), "boxes1 must be a CUDA tensor");
    TORCH_INTERNAL_ASSERT(boxes2.is_cuda(), "boxes2 must be a CUDA tensor");

    int num_boxes1 = boxes1.size(0);
    int num_boxes2 = boxes2.size(0);
    int output_size = num_boxes1 * num_boxes2;

    at::Tensor ious = at::empty({num_boxes1 * num_boxes2}, boxes1.options().dtype(at::kFloat));

    at::cuda::CUDAGuard device_guard(boxes1.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    box_iou_rotated_cuda_kernel<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
            num_boxes1, 
            num_boxes2, 
            boxes1.data_ptr<scalar_t>(),
            boxes2.data_ptr<scalar_t>(), 
            (scalar_t*)ious.data_ptr<scalar_t>(),
            mode_flag, 
            aligned
        );
    AT_CUDA_CHECK(hipGetLastError());

    // reshape from 1d array to 2d array
    auto shape = std::vector<int64_t>{num_boxes1, num_boxes2};
    return ious.view(shape);
    }
} // namespace mmrotate