#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"

__global__ void compute_iou_kernel(float* boxes1, float* boxes2, float* ious, int num_boxes) {
    // CUDA kernel implementation for IOU computation
}

extern "C" float compute_iou(float* boxes1, float* boxes2, int num_boxes) {
    // Allocate memory on GPU
    float* d_boxes1;
    float* d_boxes2;
    float* d_ious;
    hipMalloc((void**)&d_boxes1, num_boxes * sizeof(float));
    hipMalloc((void**)&d_boxes2, num_boxes * sizeof(float));
    hipMalloc((void**)&d_ious, num_boxes * sizeof(float));

    // Copy data from CPU to GPU
    hipMemcpy(d_boxes1, boxes1, num_boxes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_boxes2, boxes2, num_boxes * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    compute_iou_kernel<<<1, 1>>>(d_boxes1, d_boxes2, d_ious, num_boxes);

    // Copy result from GPU to CPU
    float iou;
    hipMemcpy(&iou, d_ious, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_boxes1);
    hipFree(d_boxes2);
    hipFree(d_ious);

    return iou;
}
